#include <iostream>
#include <fstream>
#include <cstring>
#include <hip/hip_runtime.h>
#include <iomanip>

using namespace std;

constexpr int MAX_LEN = 32; //String's Max length.
constexpr int CHAR_RANGE = 122 - 64 + 1; //String's char range
constexpr int NUM_THREADS = 256; //NUM THREAD
//65 ~ 122

__global__ void kernel_function(char* device_input, char* device_output, int N) {
    __shared__ int histogram[CHAR_RANGE]; //global historam
    __shared__ int offset[CHAR_RANGE]; //global offset
    __shared__ int count[CHAR_RANGE]; //global count

    int idx = threadIdx.x; // thread's index
    int workload = (N + NUM_THREADS - 1) / NUM_THREADS; //각 스레드가 가지는 문자열의 양.
    int start_pos = threadIdx.x * workload; // 0: 0~195 1: 196~391 //각 스레드가 시작할 위치.
    int end_pos = min(N,start_pos + workload); //각 스레드가 할 수 있는 최대 양. end_pos - 1 까지.

    for(int pos=MAX_LEN-1; pos>=0; pos--) {
        // INIT global variable
        if (idx < CHAR_RANGE) {
            histogram[idx] = 0;
            count[idx] = 0;
        }
        __syncthreads();

        int local_histogram[CHAR_RANGE] = {0,};

        for (int i=start_pos; i<end_pos; i++) {
            char now = device_input[i*MAX_LEN + pos];
            local_histogram[now-64]++;
        }
        for(int i=0; i<CHAR_RANGE; i++) atomicAdd(&histogram[i],local_histogram[i]);
        __syncthreads();

        if(idx == 0) {
            offset[0] = 0;
            for(int i=0; i<CHAR_RANGE-1; i++) offset[i+1] = offset[i] + histogram[i];
        }
        __syncthreads();
        
        for(int i=0; i<N; i++) {
            char now = device_input[i*MAX_LEN + pos];
            int index = now - 64;
            if(idx == index) {
                int after_index = offset[index] + count[index]++;
                for(int j=0; j<MAX_LEN; j++) device_output[after_index*MAX_LEN + j] = device_input[i*MAX_LEN + j];
            }
        }
        
        char* swap_temp = device_input;
        device_input = device_output;
        device_output = swap_temp;
        __syncthreads();
    }
}


void radix_sort_cuda(char* host_input, char* host_output, int N) {
    // First we have to copy these data to device.
    size_t data_size = N * MAX_LEN * sizeof(char);

    char* device_input;
    char* device_output;

    hipMalloc(&device_input, data_size);
    hipMalloc(&device_output, data_size);

    hipMemcpy(device_input, host_input, data_size, hipMemcpyHostToDevice);

    kernel_function<<<1,NUM_THREADS>>>(device_input,device_output,N);

    // and we give output to host.
    hipMemcpy(host_output,device_input,data_size,hipMemcpyDeviceToHost);
}

int main(int argc, char* argv[]) {
    int N, pos, range, ret;

    if(argc<5) {
	    cout << "Usage: " << argv[0] << " filename number_of_strings pos range" << endl;
	    return 0;
    }

    ifstream inputfile(argv[1]);

    if(!inputfile.is_open()) {
	    cout << "Unable to open file" << endl;
	    return 0;
    }

    ret=sscanf(argv[2],"%d", &N);
    if(ret==EOF || N<=0) {
	    cout << "Invalid number" << endl;
	    return 0;
    }

    ret=sscanf(argv[3],"%d", &pos);
    if(ret==EOF || pos<0 || pos>=N) {
	    cout << "Invalid position" << endl;
	    return 0;
    }

    ret=sscanf(argv[4],"%d", &range);
    if(ret==EOF || range<0 || (pos+range)>N) {
	    cout << "Invalid range" << endl;
	    return 0;
    }

    auto strArr = new char[N*MAX_LEN];
    auto output = new char[N*MAX_LEN];

    memset(strArr, 64, N * MAX_LEN);
    for (int i=0; i<N; i++) {
        inputfile >> std::setw(MAX_LEN) >> &strArr[i * MAX_LEN];
        int length = strlen(&strArr[i*MAX_LEN]);
        strArr[i*MAX_LEN + length] = 64;
    }
    inputfile.close();

    // Upper Code is the section that get data.
    radix_sort_cuda(strArr,output,N);

    cout << "\nStrings (Names) in Alphabetical order from position " << pos << ": " << "\n";
    for(int i=pos; i<N && i<(pos+range); i++) {
        cout << i << ": ";
        for(int j=0; j<MAX_LEN; j++) {
            char now = output[i*MAX_LEN+j];
            if(now != '@') cout << now;
        }
        cout << endl;
    }
        
    cout << "\n";

    delete[] strArr;

    return 0;
}
