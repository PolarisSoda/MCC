#include <iostream>
#include <fstream>
#include <cstring>
#include <hip/hip_runtime.h>

using namespace std;

constexpr int MAX_LEN = 32;
constexpr int CHAR_RANGE = 122 - 64 + 1;
constexpr int NUM_THREADS = 512;
//65 ~ 122

__global__ void kernel_function(char* device_input, char* device_output, int N) {
    //N is total amount of string.
    //we have NUM_THREADS 512
    //each THREAD HAVE 196 strings.

    __shared__ int histogram[CHAR_RANGE];
    __shared__ int offset[CHAR_RANGE];
    __shared__ int count[CHAR_RANGE];

    int idx = threadIdx.x; // thread's index
    int workload = (N + NUM_THREADS - 1) / NUM_THREADS; //각 스레드가 가지는 문자열의 양.
    int start_pos = threadIdx.x * workload; // 0: 0~195 1: 196~391 //각 스레드가 시작할 위치.
    int end_pos = min(N,start_pos + workload);

    for(int pos=MAX_LEN-1; pos>=MAX_LEN-2; pos--) {
        if (idx < CHAR_RANGE) {
            histogram[idx] = 0;
            count[idx] = 0;
        }
        __syncthreads();

        for (int i=start_pos; i<end_pos; i++) {
            char now = device_input[i * MAX_LEN + pos];
            atomicAdd(&histogram[now-64], 1);
        }
        __syncthreads();

        if(idx == 0) {
            offset[0] = 0;
            for(int i=0; i<CHAR_RANGE-1; i++) offset[i+1] = offset[i] + histogram[i];
        }
        __syncthreads();

        for(int i=0; i<N; i++) {
            char now = device_input[i*MAX_LEN + pos];
            int index = now-64;
            if(idx == index) {
                int after_index = offset[index] + count[index]++;
                for(int j=0; j<MAX_LEN; j++) device_output[after_index*MAX_LEN + j] = device_input[i*MAX_LEN + j];
            }
        }
        __syncthreads();

        if(idx == 0) {
            char* swap_temp = device_input;
            device_input = device_output;
            device_output = swap_temp;
        }
        __syncthreads();
        
    }
    //out char value is 64 ~ 123, 64 is for null values.
}


void radix_sort_cuda(char* host_input, char* host_output, int N) {

    // First we have to copy these data to device.
    size_t data_size = N * MAX_LEN * sizeof(char);

    char* device_input;
    char* device_output;

    hipMalloc(&device_input, data_size);
    hipMalloc(&device_output, data_size);

    hipMemcpy(device_input, host_input, data_size, hipMemcpyHostToDevice);

    kernel_function<<<1,NUM_THREADS>>>(device_input,device_output,N);

    // and we give output to host.
    hipMemcpy(host_output,device_output,data_size,hipMemcpyDeviceToHost);
}

int main(int argc, char* argv[]) {
    int N, pos, range, ret;

    if(argc<5) {
	    cout << "Usage: " << argv[0] << " filename number_of_strings pos range" << endl;
	    return 0;
    }

    ifstream inputfile(argv[1]);

    if(!inputfile.is_open()) {
	    cout << "Unable to open file" << endl;
	    return 0;
    }

    ret=sscanf(argv[2],"%d", &N);
    if(ret==EOF || N<=0) {
	    cout << "Invalid number" << endl;
	    return 0;
    }

    ret=sscanf(argv[3],"%d", &pos);
    if(ret==EOF || pos<0 || pos>=N) {
	    cout << "Invalid position" << endl;
	    return 0;
    }

    ret=sscanf(argv[4],"%d", &range);
    if(ret==EOF || range<0 || (pos+range)>N) {
	    cout << "Invalid range" << endl;
	    return 0;
    }

    auto strArr = new char[N*MAX_LEN];
    auto output = new char[N*MAX_LEN];

    memset(strArr,64,N*MAX_LEN);
    for(int i = 0; i < N; i++) {
        char temp_arr[MAX_LEN];
        inputfile >> temp_arr;

        int length = strlen(temp_arr);
        int pos = MAX_LEN - length;

        memcpy(&strArr[i*MAX_LEN + pos], temp_arr, length);
    }
    inputfile.close();


    // Upper Code is the section that get data.
    radix_sort_cuda(strArr,output,N);

    cout << "\nStrings (Names) in Alphabetical order from position " << pos << ": " << "\n";
    for(int i=pos; i<N && i<(pos+range); i++) {
        cout << i << ": ";
        for(int j=0; j<MAX_LEN; j++) cout << output[i*MAX_LEN + j];
        cout << endl;
    }
        
    cout << "\n";

    delete[] strArr;

    return 0;
}
