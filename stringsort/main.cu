#include <iostream>
#include <fstream>
#include <cstring>
#include <hip/hip_runtime.h>

using namespace std;

constexpr int MAX_LEN = 32;
constexpr int CHAR_RANGE = 122 - 64 + 1;
constexpr int NUM_THREADS = 512;
//65 ~ 122

__global__ void kernel_function(char* device_input, char* device_output, int N, int pos) {
    int idx = threadIdx.x;
    int workload = (N + NUM_THREADS - 1) / NUM_THREADS; //각 스레드가 가지는 문자열의 양.

    int start_pos = threadIdx.x * 196; // 0: 0~195 1: 196~391 //각 스레드가 시작할 위치.

    int end_pos = min(N,start_pos + workload);

    for (int i = start_pos; i < end_pos; i++) {
        for (int j=0; j<MAX_LEN; j++) {
            device_output[i * MAX_LEN + j] = device_input[i * MAX_LEN + j];
        }
    }
}


void radix_sort_cuda(char strArr[][MAX_LEN], int N, char output[][MAX_LEN]) {

    // First we have to copy these data to device.
    size_t data_size = N * MAX_LEN * sizeof(char);

    char* device_input;
    char* device_output;

    hipMalloc(&device_input, data_size);
    hipMalloc(&device_output, data_size);

    hipMemcpy(device_input, strArr, data_size, hipMemcpyHostToDevice);

    kernel_function<<<1,NUM_THREADS>>>(device_input,device_output,N,MAX_LEN-1);

    // and we give output to host.
    hipMemcpy(output,device_output,data_size,hipMemcpyDeviceToHost);
}

int main(int argc, char* argv[]) {
    int N, pos, range, ret;

    if(argc<5) {
	    cout << "Usage: " << argv[0] << " filename number_of_strings pos range" << endl;
	    return 0;
    }

    ifstream inputfile(argv[1]);

    if(!inputfile.is_open()) {
	    cout << "Unable to open file" << endl;
	    return 0;
    }

    ret=sscanf(argv[2],"%d", &N);
    if(ret==EOF || N<=0) {
	    cout << "Invalid number" << endl;
	    return 0;
    }

    ret=sscanf(argv[3],"%d", &pos);
    if(ret==EOF || pos<0 || pos>=N) {
	    cout << "Invalid position" << endl;
	    return 0;
    }

    ret=sscanf(argv[4],"%d", &range);
    if(ret==EOF || range<0 || (pos+range)>N) {
	    cout << "Invalid range" << endl;
	    return 0;
    }

    auto strArr = new char[N][MAX_LEN];
    auto outputs = new char[N][MAX_LEN];

    for(int i = 0; i < N; i++) {
        char temp_arr[MAX_LEN];
        inputfile >> temp_arr;

        int length = strlen(temp_arr);
        int pos = MAX_LEN - length;

        memset(strArr[i],64,MAX_LEN);
        memcpy(&strArr[i][pos], temp_arr, length);
    }
    inputfile.close();

    for(int i=0; i<N; i++) {
        for(int j=0; j<MAX_LEN; j++) {
            if(strArr[i][j] == 0) cout << 0;
            else cout << strArr[i][j];
        }
        cout << endl;
    }

    // Upper Code is the section that get data.
    radix_sort_cuda(strArr,N,outputs);

    cout << "\nStrings (Names) in Alphabetical order from position " << pos << ": " << "\n";
    for(int i=0; i<N; i++) {
        cout << i << ": ";
        for(int j=0; j<MAX_LEN; j++) cout << outputs[i*MAX_LEN +j];
        cout << endl;
    }
        
    cout << "\n";

    delete[] strArr;

    return 0;
}
